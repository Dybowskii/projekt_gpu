#include "hip/hip_runtime.h"
﻿#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
using namespace std;

__constant__ int sharpen[9] = {
    0, -1, 0, 
    -1, 5, -1, 
    0, -1, 0
};
__constant__ int ridge[9] = {
	-1, -1, -1,
	-1, 8, -1,
	-1, -1, -1
};


__global__ void kernel(unsigned char* input, unsigned char* output,
	int width, int height, int offset, int total_pixels)
{
	int index = threadIdx.x + offset;
	int filter_index = 0;
	int pixel_value = 0;
	int x = index % width;
	int y = index / width;
	if (index < total_pixels) {
		if (x == 0 || x == width - 1 || y == 0 || y == height - 1) {
			output[index] = input[index];
			return;
		}
		for (int i = -1; i<= 1; i++)
			for (int j = -1; j <= 1; j++)
			{
				int neighbor_index = (y + j) * width + (x + i);
				pixel_value += input[neighbor_index] * sharpen[filter_index];
				filter_index++;
			}
		if (pixel_value <= 0)
		{
			output[index] = 0;
		}
			
		else if (pixel_value > 255)
		{
			output[index] = 255;
		}
		else
		{
			output[index] = pixel_value;
		}
			
	}
}

int main()
{
	int width, height, channels;
	unsigned char* input_cpu = stbi_load("image.png", &width, &height, &channels, 1);
	int total_pixels = width * height;
	unsigned char* input_gpu, * output_gpu;
	hipMalloc(&input_gpu, total_pixels);
	hipMalloc(&output_gpu, total_pixels);
	hipMemcpy(input_gpu, input_cpu, total_pixels, hipMemcpyHostToDevice);
	
	cout << total_pixels << endl;
	int temp = 1;
	for (int i = 0; i < total_pixels; i += 1024) {
		kernel << <1, 1024 >> > (input_gpu, output_gpu, width, height, i, total_pixels);
		hipDeviceSynchronize();
	}
	unsigned char* output_cpu = new unsigned char[total_pixels];
	hipMemcpy(output_cpu, output_gpu, total_pixels, hipMemcpyDeviceToHost);
	stbi_write_png("output_image.png", width, height, 1, output_cpu, width);
	return 0;
}